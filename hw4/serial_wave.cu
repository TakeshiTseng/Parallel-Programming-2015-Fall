/**********************************************************************
 * DESCRIPTION:
 *   Serial Concurrent Wave Equation - C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265
#define FAC 6.2831853
#define SQTAU 0.09

void check_param(void);
void init_line(void);
void update (void);
void printfinal (void);
__global__ void cu_do_match();
int nsteps,                 	/* number of time steps */
    tpoints, 	     		/* total points along string */
    rcode;                  	/* generic return code */
float values[MAXPOINTS+2], 	/* values at time t */
      oldval[MAXPOINTS+2], 	/* values at time (t-dt) */
      newval[MAXPOINTS+2]; 	/* values at time (t+dt) */


/**********************************************************************
 * Checks input values from parameters
 *********************************************************************/
void check_param(void)
{
    char tchar[20];

    /* check number of points, number of iterations */
    while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
        printf("Enter number of points along vibrating string [%d-%d]: "
                ,MINPOINTS, MAXPOINTS);
        scanf("%s", tchar);
        tpoints = atoi(tchar);
        if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
            printf("Invalid. Please enter value between %d and %d\n", 
                    MINPOINTS, MAXPOINTS);
    }
    while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
        printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
        scanf("%s", tchar);
        nsteps = atoi(tchar);
        if ((nsteps < 1) || (nsteps > MAXSTEPS))
            printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
    }

    printf("Using points = %d, steps = %d\n", tpoints, nsteps);

}

/**********************************************************************
 *     Initialize points on line
 *********************************************************************/
void init_line(void)
{
    int i, j;
    float x, k, tmp;

    /* Calculate initial values based on sine curve */
    k = 0.0; 
    tmp = tpoints - 1;
    for (j = 1; j <= tpoints; j++) {
        x = k/tmp;
        values[j] = sin (FAC * x);
        k = k + 1.0;
    }

    /* Initialize old values array */
    for (i = 1; i <= tpoints; i++) 
        oldval[i] = values[i];
}

/**********************************************************************
 *      Calculate new values using wave equation
 *********************************************************************/
__global__ void cu_do_match(float* nv, float* ov, float* v, int tpoints)
{
    int i = threadIdx.x;
    if(i == 1 || i == tpoints) {
        nv[i] = 0.0;
    } else {
        nv[i] = (2.0 * v[i]) - ov[i] + (SQTAU *  (-2.0) * v[i]);
    }
    ov[i] = v[i];
    v[i] = nv[i];
}


/**********************************************************************
 *     Update all values along line a specified number of times
 *********************************************************************/
void update()
{
    int i;
    float* nv;
    float* ov;
    float* v;

    hipMalloc((void**) &nv, MAXPOINTS + 2);
    hipMalloc((void**) &ov, MAXPOINTS + 2);
    hipMalloc((void**) &v, MAXPOINTS + 2);
    hipMemcpy(nv, newval, sizeof(float)*tpoints, hipMemcpyHostToDevice);
    hipMemcpy(ov, oldval, sizeof(float)*tpoints, hipMemcpyHostToDevice);
    hipMemcpy(v, values, sizeof(float)*tpoints, hipMemcpyHostToDevice);

    /* Update values for each time step */
    for (i = 1; i<= nsteps; i++) {
        /* Update points along line for this time step */
        cu_do_match<<<1, tpoints>>>(nv, ov, v, tpoints);
    }
    hipMemcpy(newval, nv, sizeof(float)*tpoints, hipMemcpyDeviceToHost);
    hipMemcpy(oldval, ov, sizeof(float)*tpoints, hipMemcpyDeviceToHost);
    hipMemcpy(values, v, sizeof(float)*tpoints, hipMemcpyDeviceToHost);

}

/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal()
{
    int i;

    for (i = 1; i <= tpoints; i++) {
        printf("%6.4f ", values[i]);
        if (i%10 == 0)
            printf("\n");
    }
}

/**********************************************************************
 *	Main program
 *********************************************************************/
int main(int argc, char *argv[])
{
    sscanf(argv[1],"%d",&tpoints);
    sscanf(argv[2],"%d",&nsteps);
    check_param();
    printf("Initializing points on the line...\n");
    init_line();
    printf("Updating all points for all time steps...\n");
    update();
    printf("Printing final results...\n");
    printfinal();
    printf("\nDone.\n\n");

    return 0;
}
